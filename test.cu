
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <fstream>
#include <iterator>
#include <fstream>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__
void saxpy(int n, float a, float *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  for (int j=0; j < 10000000; j++){
    if (i < n) {
      x[i] += 1;
    }
  }
}

int main(void) {
  std::chrono::time_point<std::chrono::system_clock> start, end;
  start = std::chrono::system_clock::now();
  float *x, *d_x;

  int N = 10000000;
  hipHostAlloc((void**)&x, N*sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);

  int n_gpus=4;
  for (int i = 0; i < N/n_gpus; i++) {
      x[i] = 1.0f;
  }

  hipStream_t streams[n_gpus];
  #pragma omp parallel num_threads(n_gpus)
  {
    int d=omp_get_thread_num();
    std::cout << "Thread" << d << "activated\n";
    std::cout << d << " Device initialized \n";
    int start_idx = (N/n_gpus)*omp_get_thread_num();
    int end_idx = start_idx + (N/n_gpus);
    std::cout << "Start index: " << start_idx << "\n";
    std::cout << "End index: " << end_idx << "\n";
    hipSetDevice(omp_get_thread_num());
    hipStreamCreate(&streams[d]);

    hipMalloc(&d_x, (N/n_gpus)*sizeof(float));
    hipMemcpyAsync(d_x, x+start_idx, (N/n_gpus)*sizeof(float), hipMemcpyHostToDevice, streams[d]);

    saxpy<<<(N+127)/128, 128, 0, streams[d]>>>(N/n_gpus, 2.0f, d_x);

    hipMemcpyAsync(x+start_idx, d_x, (N/n_gpus)*sizeof(float), hipMemcpyDeviceToHost, streams[d]);
    hipDeviceSynchronize();
  }

  hipError_t err = hipGetLastError();  // add
  if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
  std::cout << x[0] << "\n";

  // measure elapsed kernal runtime
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;
  std::time_t end_time = std::chrono::system_clock::to_time_t(end);
  std::cout << "Elapsed Time: " << elapsed_seconds.count() << "s\n";

  return;
}