#include <stdio.h>
#include <iostream>
#include <chrono>
#include <fstream>
#include <iterator>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  for (int j=0; j < 1000; j++){
    if (i < n) {
      float num = x[i];
      y[i] = 20*i;
    }
  }
}

extern "C" {
float* save_arr()
  {
    int N = 1<<20;
    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
      x[i] = 1.0f;
      y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

    hipDeviceSynchronize();

    // measure elapsed kernal runtime
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;
    std::time_t end_time = std::chrono::system_clock::to_time_t(end);
    std::cout << "Elapsed Time: " << elapsed_seconds.count() << "s\n";

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    return y;
  }
}